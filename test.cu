#include "hip/hip_runtime.h"
#include "stdio.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"


#define BLOCKS 32


__global__ void setup_kernel(hiprandState *state, unsigned long seed)
{
    printf("hhh");
	// int tid = blockIdx.x *blockDim.x + threadIdx.x; //获取线程号0~blocks*THREAD_NUM-1  grid划分成1维，block划分为1维
	// hiprand_init(seed, tid, 0, &state[tid]);// initialize the state
    
}

__global__ void use(hiprandState *globalState)
{
    
	// unsigned int j;
	// int tid = blockIdx.x *blockDim.x + threadIdx.x; //获取线程号0~blocks*THREAD_NUM-1  grid划分成1维，block划分为1维
	// hiprandState localState = globalState[tid];
	// j = (hiprand(&localState));
	// printf("%u\n", j);
    printf("hhh");
}

int main()
{
	hiprandState* devStates;  //创建一个随机算法状态的对象
	hipMalloc(&devStates, BLOCKS * THREAD_NUM * sizeof(hiprandState));
	srand(time(0));

	setup_kernel << <BLOCKS, THREAD_NUM >> > (devStates, rand()); // blocks number is POP. thread number is P
	use << < BLOCKS, THREAD_NUM >> > (devStates);
    
    return 0;
}

